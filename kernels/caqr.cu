
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <stdbool.h>
#include <assert.h>
#include <sys/time.h>
#include <hip/hip_runtime_api.h>

//#define ENABLE_MAGMA

#ifdef ENABLE_MAGMA
#include "magma.h"
#endif

//Scalar type and panel width
#define Scalar float

//PR is how big the square trailing update block matrix should be (per CUDA block)
//(PR^2 + 2 * PR * PC) * sizeof(Scalar) should fit in 48 KiB
#define PR 64
//PC is how many columns of A get grouped into one compressed block Householder transform
#define PC 4
//Trials for timing
#define trials 3

//integer division a/b, rounded up
#define ceildiv(a, b) ((a) / (b) + ((a) % (b) != 0))

void dgemm(Scalar* A, Scalar* B, Scalar* C, int k, int m, int n);
void explicitQR(Scalar* A, Scalar* tau, Scalar* Q, Scalar* R, int m, int n);
void identity(Scalar* A, int m);

//print a column-major matrix row-by-row (for debugging)
__host__ __device__ void printMat(Scalar* mat, int m, int n)
{
  printf("Matrix %d x %d, row by row:\n", m, n);
  for(int i = 0; i < m; i++)
  {
    for(int j = 0; j < n; j++)
    {
      printf("%9f ", mat[j * m + i]);
    }
    printf("\n");
  }
  printf("\n");
}

void getPanelDims(int m, int n, int* rowPanels, int* colPanels)
{
  *colPanels = ceildiv(n, PC);
  *rowPanels = 1;
  if(m > PR)
    *rowPanels += ceildiv(m - PR, PR - PC);
}

//Do block Householder factorization of the first PC columns of mat, starting at pc
//this kernel is only meant to be run on a single block with as many threads as possible
//this is because it contains many synchronization points
__global__ void panelHouseholderKernel(Scalar* mat, Scalar* tau, Scalar* W, int m, int n, int pr, int pc)
{
  //useful to think of shared memory buffer as a stack
  //for simple dynamic allocations
  extern __shared__ Scalar sharedBuf[];
  //Preallocate all shared arrays here
  Scalar* toReduce = sharedBuf;
  Scalar* finalReduce = sharedBuf + blockDim.x;
  int finalReduceNum = 16;
  Scalar* panel = finalReduce + finalReduceNum;
  Scalar* Wshared = panel + (PR * PC);
  Scalar* Acol = Wshared + (PR * PC);
  //zero out Wshared
  for(int i = 0; i < PR * PC; i += blockDim.x)
  {
    int index = i + threadIdx.x;
    if(index < PR * PC)
      Wshared[index] = 0;
  }
  //load panel into shared
  for(int i = 0; i < PR * PC; i += blockDim.x)
  {
    int index = i + threadIdx.x;
    if(index < PR * PC)
    {
      int col = index / PR;
      int row = index % PR;
      Scalar matVal = 0;
      if(pc + col < n && pr + row >= 0)
        matVal = mat[(pc + col) * m + pr + row];
      panel[row + col * PR] = matVal;
    }
  }
  __syncthreads();
  /*
  if(threadIdx.x == 0)
  {
    printf("PANEL LOCATION: %d, %d\n", pr, pc);
    printf("PANEL BEFORE FACTORIZATION\n");
    printMat(panel, PR, PC);
  }
  */
  for(int col = 0; col < PC && pc + col < n; col++)
  {
    //is the panel at the bottom of A?
    bool bottomPanel = pr == m - PR;
    //does col 0 of panel cross A's diagonal?
    bool topPanel = pr < pc;
    int vstart;
    int vend;
    if(topPanel && bottomPanel)
    {
      vstart = pc - pr + col;
      vend = PR;
    }
    else if(!topPanel && bottomPanel)
    {
      vstart = col;
      vend = PR;
    }
    else if(topPanel && !bottomPanel)
    {
      //vstart needs to be at or below A's diagonal, even if
      //panel boundaries extends above it
      vstart = pc - pr + col;
      vend = PR - PC + col + 1;
    }
    else
    {
      //neither top nor bottom panel
      vstart = col;
      vend = PR - PC + col + 1;
    }
    int vlen = vend - vstart;
    //note: when computing tau/reflectors,
    //work directly with global mat (only 2 flops per element anyway)
    //compute the inner product and norm of column
    Scalar innerProd = 0;
    {
      Scalar localInnerProd = 0;
      //use a cyclic row distribution for perfect coalesced accesses
      for(int i = vstart; i < vend; i += blockDim.x)
      {
        int index = i + threadIdx.x;
        if(index < vend)
        {
          localInnerProd += panel[index + col * PR] * panel[index + col * PR];
        }
      }
      //now, sum up the localInnerProds across the whole block
      //write the partial sums to shared, then do a simple linear reduction
      toReduce[threadIdx.x] = localInnerProd;
      __syncthreads();
      if(threadIdx.x < finalReduceNum)
      {
        localInnerProd = 0;
        for(int i = 0; i < blockDim.x; i += finalReduceNum)
        {
          int index = i + threadIdx.x;
          if(index < blockDim.x)
            localInnerProd += toReduce[index];
        }
        finalReduce[threadIdx.x] = localInnerProd;
      }
      __syncthreads();
      //now, every thread sums up finalReduce to get innerProd
      for(int i = 0; i < finalReduceNum; i++)
        innerProd += finalReduce[i];
    }
    Scalar leading = panel[col * PR + vstart];
    Scalar norm = sqrt(innerProd);
    Scalar sign = (leading < 0) ? -1.0 : 1.0;
    Scalar u = leading + sign * norm;
    Scalar thisTau = sign * u / norm;
    /*
    if(threadIdx.x == 0)
    {
      printf("BMK\n");
      printf("Leading entry: %f\n", leading);
      printf("norm: %f\n", norm);
      printf("tau: %f\n", thisTau);
    }
    */
    //compute entire w vector in-place, storing it back to A subdiag
    for(int i = vstart; i < vend; i += blockDim.x)
    {
      int index = i + threadIdx.x;
      if(index == vstart)
      {
        //thread 0 uniquely responsible for setting R diagonal entry and tau
        tau[col] = thisTau;
        panel[col * PR + vstart] = -sign * norm;
      }
      else if(index < vend)
      {
        panel[col * PR + index] /= u;
      }
    }
    __syncthreads();
    //v is now fully computed and stored back to panel
    //compute z vector using W,Y
    //each thread will compute one entry in z
    for(int i = 0; i < PR; i += blockDim.x)
    {
      int index = i + threadIdx.x;
      if(index < PR)
      {
        Scalar zval = 0;
        //set zval to v[index]
        if(index == vstart)
          zval = -thisTau;
        else if(index > vstart && index < vend)
          zval = -thisTau * panel[col * PR + index];
        //finish computing entry i of z
        //compute zval as (W * Y^T * v)(i)
        Scalar wytvi = 0;
        for(int j = 0; j < PR; j++)
        {
          //need inner product of row i of W and row j of Y
          //this is (WY^T)(i, j)
          //use the fact that only the first col+1 columns of W and Y are nonzero
          Scalar wyt = 0;
          for(int k = 0; k < col; k++)
          {
            Scalar yval = 0;
            //find the set of rows for column k of panel
            int vstartK, vendK;
            if(topPanel && bottomPanel)
            {
              vstartK = pc - pr + k;
              vendK = PR;
            }
            else if(!topPanel && bottomPanel)
            {
              vstartK = k;
              vendK = PR;
            }
            else if(topPanel && !bottomPanel)
            {
              //vstart needs to be at or below A's diagonal, even if
              //panel boundaries extends above it
              vstartK = pc - pr + k;
              vendK = PR - PC + k + 1;
            }
            else
            {
              //neither top nor bottom panel
              vstartK = k;
              vendK = PR - PC + k + 1;
            }
            if(j > vstartK && j < vendK)
              yval = panel[k * PR + j];
            else if(j == vstartK)
              yval = 1;
            wyt += Wshared[k * PR + index] * yval;
          }
          Scalar vval = 0;
          if(j == vstart)
            vval = 1;
          else if(j > vstart && j < vend)
            vval = panel[col * PR + j];
          wytvi += wyt * vval;
        }
        zval -= thisTau * wytvi;
        Wshared[col * PR + index] = zval;
      }
    }
    __syncthreads();
    //apply reflector in col to remaining columns in panel
    for(int applyCol = col + 1; applyCol < PC && pc + applyCol < n; applyCol++)
    {
      //Create a copy of the updating column of A which will
      //persist while each entry is computed
      //Only the height range [vstart, m) is read, used and written back
      for(int i = 0; i < vlen; i += blockDim.x)
      {
        int index = i + threadIdx.x;
        if(index < vlen)
          Acol[index] = panel[applyCol * PR + vstart + index];
      }
      __syncthreads();
      for(int applyRow = vstart; applyRow < vend; applyRow += blockDim.x)
      {
        int index = applyRow + threadIdx.x;
        if(index < vend)
        {
          Scalar val = Acol[index - vstart];
          Scalar vIndex = 0;
          if(index == vstart)
            vIndex = 1;
          else
            vIndex = panel[col * PR + index];
          for(int i = vstart; i < vend; i++)
          {
            Scalar vi = 0;
            if(i == vstart)
              vi = 1;
            else
              vi = panel[col * PR + i];
            val -= thisTau * vIndex * vi * Acol[i - vstart];
          }
          panel[applyCol * PR + index] = val;
        }
      }
    }
  }
  __syncthreads();
  //write out W and panel back to global
  for(int i = 0; i < PR * PC; i += blockDim.x)
  {
    int index = i + threadIdx.x;
    if(index < PR * PC)
    {
      W[index] = Wshared[index];
    }
  }
  for(int i = 0; i < PR * PC; i += blockDim.x)
  {
    int index = i + threadIdx.x;
    if(index < PR * PC)
    {
      int row = index % PR;
      int col = index / PR;
      mat[pr + row + (pc + col) * m] = panel[row + col * PR];
    }
  }
  /*
  if(threadIdx.x == 0)
  {
    printf("PANEL AFTER FACTORIZATION\n");
    printMat(panel, PR, PC);
  }
  */
}

__global__ void trailingUpdateKernel(Scalar* mat, Scalar* W, int m, int n, int pr, int pc)
{
  //All dynamic shared memory goes here
  //Is a flat 48 KiB buffer, free for use by each block
  extern __shared__ Scalar sharedBuf[];
  //determine range of rows "owned" by this thread
  //Allocate some shared arrays that all blocks will use for computations
  //Note: W is not transposed in memory (coalesce memory accesses)
  //The YW^T entries are computed as inner products of rows of Yblock and Wblock
  Scalar* Wshared = &sharedBuf[0];
  Scalar* Y = &Wshared[PR * PC];
  Scalar* Acol = &Y[PR * PC];
  //is the panel at the bottom of A?
  bool bottomPanel = pr == m - PR;
  //does col 0 of panel cross A's diagonal?
  bool topPanel = pr <= pc;
  //update trailing columns of A: A = (I + YW^T)A
  //Each block reads into Wblock/Yblock/Ablock, does multiplication and writes results out to Ascratch
  int blockCol = pc + PC + blockIdx.x * PR;
  //first load in Y block
  //it will stay constant for whole kernel
  for(int i = 0; i < PR * PC; i += blockDim.x)
  {
    int index = i + threadIdx.x;
    if(index < PR * PC)
    {
      int row = index % PR;
      int col = index / PR;
      int vstart;
      int vend;
      if(topPanel && bottomPanel)
      {
        vstart = pc - pr + col;
        vend = PR;
      }
      else if(!topPanel && bottomPanel)
      {
        vstart = col;
        vend = PR;
      }
      else if(topPanel && !bottomPanel)
      {
        //vstart needs to be at or below A's diagonal, even if
        //panel boundaries extends above it
        vstart = pc - pr + col;
        vend = PR - PC + col + 1;
      }
      else
      {
        //neither top nor bottom panel
        vstart = col;
        vend = PR - PC + col + 1;
      }
      if(!bottomPanel)
        vend = PR - PC + col + 1;
      int matRow = pr + row;
      int matCol = pc + col;
      //Y's columns are simply the reflectors stored in mat's subdiagonal.
      //this reads back the implicit 0/1 entries
      Scalar yval = 0;
      if(matRow < m && matCol < n)
      {
        if(row > vstart && row < vend)
          yval = mat[matRow + m * matCol];
        else if(row == vstart)
          yval = 1;
      }
      Y[row + col * PR] = yval;
    }
  }
  //load Wblock into shared (from the global W)
  for(int i = 0; i < PR * PC; i += blockDim.x)
  {
    int index = i + threadIdx.x;
    if(index < PR * PC)
    {
      Wshared[index] = W[index];
    }
  }
  /*
  __syncthreads();
  if(threadIdx.x == 0)
  {
    printf("Y matrix for updating trail of %d, %d\n", pr, pc);
    printMat(Y, PR, PC);
    printf("W matrix for updating trail:\n");
    printMat(Wshared, PR, PC);
  }
  */
  __syncthreads();
  //For each column to update...
  for(int applyCol = 0; applyCol < PR && applyCol + pc + PC < n; applyCol++)
  {
    //Save a copy of the column to Acol
    for(int j = 0; j < PR; j += blockDim.x)
    {
      int index = j + threadIdx.x;
      if(index < PR)
      {
        if(pr + index < m && blockCol + applyCol < n)
          Acol[index] = mat[pr + index + (blockCol + applyCol) * m];
        else
          Acol[index] = 0;
      }
    }
    __syncthreads();
    //Compute the updated (I + Y * W^T) * Acol
    for(int i = 0; i < PR; i += blockDim.x)
    {
      int entry = i + threadIdx.x;
      if(entry < PR)
      {
        //"entry" is the index of entry of new Acol being computed
        Scalar val = Acol[entry];
        for(int j = 0; j < PR; j++)
        {
          Scalar ywt = 0;
          for(int k = 0; k < PC; k++)
          {
            ywt += Y[entry + k * PR] * Wshared[j + k * PR];
          }
          val += ywt * Acol[j];
        }
        //can safely write this back immediately
        if(pr + entry < m && blockCol + applyCol < n)
          mat[pr + entry + (blockCol + applyCol) * m] = val;
      }
    }
    __syncthreads();
  }
}

#define HANDLE_ERROR(x) \
{\
  hipError_t err = x; \
  if(x) {printf("CUDA error on line %i: %d\n", __LINE__, x); exit(1);} \
}

//Host wrapper for the main CUDA kernel
//No extra overhead since copies to/from device would be necessary anyway
void mmqr(Scalar* mat, Scalar* tau, int m, int n)
{
  int rowPanels, colPanels;
  getPanelDims(m, n, &rowPanels, &colPanels);
  hipDeviceProp_t prop;
  HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));
  int shmem = prop.sharedMemPerBlock;
  int factorThreads = 256;
  //int sm = prop.multiProcessorCount;
  //MUST have at least 48 KiB of shared memory for this to work in its current state
  //TODO: adapt to any amount of shared
  if(prop.sharedMemPerBlock < 48 * 1024)
  {
    //this should never actually happen
    puts("CUDA device has < 48 KiB shared memory per block!");
    exit(1);
  }
  Scalar* Adev;
  HANDLE_ERROR(hipMalloc((void**) &Adev, m * n * sizeof(Scalar)));
  Scalar* W;
  hipMalloc((void**) &W, PR * PC * sizeof(Scalar));
  Scalar* tauDev;
  HANDLE_ERROR(hipMalloc((void**) &tauDev, rowPanels * colPanels * n * sizeof(Scalar)));
  HANDLE_ERROR(hipMemcpy(Adev, mat, m * n * sizeof(Scalar), hipMemcpyHostToDevice));
  //launch the kernel
  //
  //only use one block and fixed threads for main kernel,
  //but main kernel will itself launch several blocks to saturate FLOPs during trailing updates)
  //want one block per SM and as many threads as possible (up to 1 per row of A)
  int pcCount = 0;
  for(int pc = 0; pc < n; pc += PC)
  {
    int prCount = 0;
    for(int pr = m - PR; pr + PR > pc && pr >= 0; pr -= (PR-PC))
    {
      //know exactly how much shared memory each kernel needs (at runtime)
      int kernel1shared = (factorThreads + 16 + 2 * PR * PC + PR) * sizeof(Scalar);
      assert(kernel1shared <= shmem);
      /*
      printf("Kernel 1 (panel factor) needs %d bytes shared\n", kernel1shared);
      printf("Launching kernel 1...");
      */
      Scalar* panelTau = &tauDev[(rowPanels * pcCount + prCount) * PC];
      panelHouseholderKernel<<<1, factorThreads, kernel1shared>>>(Adev, panelTau, W, m, n, pr, pc);
      /*
      HANDLE_ERROR(cudaMemcpy(mat, Adev, m * n * sizeof(Scalar), cudaMemcpyDeviceToHost));
      printf("Full matrix after processing panel %d, %d:\n", pr, pc);
      printMat(mat, m, n);
      puts("done");
      */
      int changedColumns = PC;
      if(changedColumns + pc > n)
        changedColumns = n - pc;
      if(pc + PC < n)
      {
        int blocks = ceildiv(n - pc - PC, PR);
        int kernel2shared = (2 * PR * PC + PR) * sizeof(Scalar);
        assert(kernel2shared <= shmem);
        /*
        printf("Kernel 2 (trailing update) needs %d bytes shared\n", kernel2shared);
        printf("Launching kernel 2...");
        */
        trailingUpdateKernel<<<blocks, 512, kernel2shared>>>(Adev, W, m, n, pr, pc);
        //puts("done");
        //HANDLE_ERROR(cudaMemcpy(mat, Adev, m * n * sizeof(Scalar), cudaMemcpyDeviceToHost));
        //printf("After trailing update, full matrix:\n");
        //printMat(mat, m, n);
      }
      prCount++;
    }
    pcCount++;
  }
  //retrieve A and tau
  HANDLE_ERROR(hipMemcpy(mat, Adev, m * n * sizeof(Scalar), hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(tau, tauDev, rowPanels * colPanels * PC * sizeof(Scalar), hipMemcpyDeviceToHost));
  hipFree(W);
  hipFree(tauDev);
  hipFree(Adev);
}

#ifdef ENABLE_MAGMA
void magmaQR(Scalar* mat, Scalar* tau, int m, int n)
{
  Scalar* Adev;
  HANDLE_ERROR(cudaMalloc((void**) &Adev, m * n * sizeof(Scalar)));
  HANDLE_ERROR(cudaMemcpy(Adev, mat, m * n * sizeof(Scalar), cudaMemcpyHostToDevice));
  magma_int_t info;
  magma_sgeqrf2_gpu(m, n, Adev, m, tau, &info);
  HANDLE_ERROR(cudaMemcpy(mat, Adev, m * n * sizeof(Scalar), cudaMemcpyDeviceToHost));
}
#endif

//A = mxm identity matrix
void identity(Scalar* A, int m)
{
  for(int i = 0; i < m * m; i++)
    A[i] = 0;
  for(int i = 0; i < m; i++)
  {
    A[i + m * i] = 1;
  }
}

//From A and tau array produced by mmqr,
//explicitly find Q and R matrices
//Q is mxm, A and R are mxn
//All matrices are column-major
void explicitQR(Scalar* A, Scalar* tau, Scalar* Q, Scalar* R, int m, int n)
{
  //first, R is simply the upper triangular part of A (including diagonal)
  for(int i = 0; i < n; i++)
  {
    for(int j = 0; j < m; j++)
    {
      if(i >= j)
        R[i * m + j] = A[i * m + j];
      else
        R[i * m + j] = 0;
    }
  }
  //next, Q is the result of applying each Householder reflector
  //(stored in subdiagonals) to I(m)
  //note: this is very expensive to do naively on host
  //first, get I(m) into Q
  identity(Q, m);
  int rowPanels, colPanels;
  getPanelDims(m, n, &rowPanels, &colPanels);
  int pcCount = 0;
  for(int pc = 0; pc < n; pc += PC)
  {
    //then bottom to top, sliding panel up by R-C each iteration
    //prCount gives row index of panel (bottom is 0)
    int prCount = 0;
    for(int pr = m - PR; (pr + PR > pc) && pr >= 0; pr -= (PR-PC))
    {
      //is the panel at the bottom of A?
      bool bottomPanel = pr == m - PR;
      //does col 0 of panel cross A's diagonal?
      bool topPanel = pr <= pc;
      for(int col = 0; col < PC && col + pc < n; col++)
      {
        Scalar tauVal = tau[(rowPanels * pcCount + prCount) * PC + col];
        //update each trailing column (pr:pr+R, pc+C:N):
        //for each column, compute HH reflectors
        //(middle panels are both top and bottom)
        int vstart;
        int vend;
        if(topPanel && bottomPanel)
        {
          vstart = pc - pr + col;
          vend = PR;
        }
        else if(!topPanel && bottomPanel)
        {
          vstart = col;
          vend = PR;
        }
        else if(topPanel && !bottomPanel)
        {
          //vstart needs to be at or below A's diagonal, even if
          //panel boundaries extends above it
          vstart = pc - pr + col;
          vend = PR - PC + col + 1;
        }
        else
        {
          //neither top nor bottom panel
          vstart = col;
          vend = PR - PC + col + 1;
        }
        Scalar* v = (Scalar*) malloc(m * sizeof(Scalar));
        //read v from subdiagonal of A
        for(int i = 0; i < m; i++)
        {
          if(i < pr + vstart || i >= pr + vend)
            v[i] = 0;
          else if(i == pr + vstart)
            v[i] = 1;
          else
            v[i] = A[(pc + col) * m + i];
        }
        // printf("REFLECTOR: column %d in panel %d, %d:\n", col, pr, pc);
        // for(int i = 0; i < m; i++)
        // {
        //   printf("%9f ", v[i]);
        // }
        // putchar('\n');
        //create H matrix for this reflector
        Scalar* H = (Scalar*) malloc(m * m * sizeof(Scalar));
        identity(H, m);
        for(int j = 0; j < m; j++)
        {
          for(int k = 0; k < m; k++)
          {
            H[k + j * m] -= tauVal * v[k] * v[j];
          }
        }
        //dgemm can't multiply Q by H in-place,
        //so make a persistent copy of Q
        Scalar* prevQ = (Scalar*) malloc(m * m * sizeof(Scalar));
        for(int j = 0; j < m * m; j++)
          prevQ[j] = Q[j];
        dgemm(prevQ, H, Q, m, m, m);
        free(prevQ);
        free(v);
        free(H);
      }
      prCount++;
    }
    pcCount++;
  }
}

//General dense matrix-matrix product
//A is kxm, B is mxn and C is kxn
//All matrices are column-major
void dgemm(Scalar* A, Scalar* B, Scalar* C, int k, int m, int n)
{
  for(int i = 0; i < n; i++)
  {
    for(int j = 0; j < k; j++)
    {
      //compute C(j, i) as
      //row j of A dotted with column i of B
      Scalar cval = 0;
      for(int l = 0; l < m; l++)
      {
        cval += A[j + l * k] * B[l + i * m];
      }
      C[j + i * k] = cval;
    }
  }
}

// int main(int argc, const char** argv)
// {
//   HANDLE_ERROR(cudaSetDevice(0));
// #ifdef ENABLE_MAGMA
//   magma_init();
// #endif
//   if(argc < 3)
//   {
//     puts("Usage: ./qr_device m n");
//     exit(1);
//   }
//   int m = atoi(argv[1]);
//   int n = atoi(argv[2]);
//   //make m,n fit to panels
//   {
//     int numPanels = ((double) (m - PR) / (PR - PC) + 0.5);
//     m = PR + numPanels * (PR - PC);
//   }
//   {
//     int numPanels = ((double) n / PC + 0.5);
//     if(numPanels == 0)
//       numPanels = 1;
//     n = numPanels * PC;
//     while(n > m)
//       n -= PC;
//   }
//   printf("Exact problem size: %dx%d\n", m, n);
//   assert(m && n && m >= n);
//   //only use one device (at least, for now)
//   //First, make sure device is using proper 48 KB of shared, 16 KB L1
//   //during all calls to L1 kernel
//   //Note that this is not the default
//   HANDLE_ERROR(cudaDeviceSetCacheConfig(cudaFuncCachePreferShared));
//   cudaDeviceProp prop;
//   HANDLE_ERROR(cudaGetDeviceProperties(&prop, 0));
//   int sm = prop.multiProcessorCount;
//   printf("Testing mmqr on \"%s\"\n", prop.name);
//   printf("Device has %d SMs, %zu bytes of shared, and up to %d threads per block\n", sm, prop.sharedMemPerBlock, prop.maxThreadsPerBlock);
//   if(sizeof(Scalar) == 4)
//   {
//     HANDLE_ERROR(cudaDeviceSetSharedMemConfig(cudaSharedMemBankSizeFourByte));
//   }
//   else if(sizeof(Scalar) == 8)
//   {
//     HANDLE_ERROR(cudaDeviceSetSharedMemConfig(cudaSharedMemBankSizeEightByte));
//   }
//   else
//   {
//     puts("Only float (32-bit) and double (64-bit) reals are supported scalar types");
//     exit(1);
//   }
//   Scalar* A = (Scalar*) malloc(m * n * sizeof(Scalar));
//   Scalar* RV = (Scalar*) malloc(m * n * sizeof(Scalar));
//   int rowPanels, colPanels;
//   getPanelDims(m, n, &rowPanels, &colPanels);
//   Scalar* tau = (Scalar*) malloc(rowPanels * colPanels * PC * sizeof(Scalar));
//   srand(12);
//   //initialize A randomly
//   for(int i = 0; i < m * n; i++)
//   {
//     A[i] = (Scalar) rand() / RAND_MAX;
//     RV[i] = A[i];
//   }
//   //puts("A matrix:\n");
//   //printMat(A, m, n);
//   double mmqrElapsed = 0;
//   struct timeval currentTime;
//   gettimeofday(&currentTime, NULL);
//   for(int i = 0; i < trials; i++)
//   {
//     mmqr(RV, tau, m, n);
//     struct timeval nextTime;
//     gettimeofday(&nextTime, NULL);
//     //add to elapsed time
//     mmqrElapsed += (nextTime.tv_sec + 1e-6 * nextTime.tv_usec) - (currentTime.tv_sec + 1e-6 * currentTime.tv_usec);
//     currentTime = nextTime;
//     //refresh RV for next trial (this isn't part of the algorithm and so isn't timed)
//     if(i != trials - 1)
//       memcpy(RV, A, m * n * sizeof(Scalar));
//   }
//   printf(" MMQR ran QR on %dx%d matrix in %f s (avg over %d)\n", m, n, mmqrElapsed / trials, trials);
// #ifdef ENABLE_MAGMA
//   double magmaElapsed = 0;
//   gettimeofday(&currentTime, NULL);
//   for(int i = 0; i < trials; i++)
//   {
//     magmaQR(RV, tau, m, n);
//     struct timeval nextTime;
//     gettimeofday(&nextTime, NULL);
//     //add to elapsed time
//     magmaElapsed += (nextTime.tv_sec + 1e-6 * nextTime.tv_usec) - (currentTime.tv_sec + 1e-6 * currentTime.tv_usec);
//     currentTime = nextTime;
//     //refresh RV for next trial (this isn't part of the algorithm and so isn't timed)
//     if(i != trials - 1)
//       memcpy(RV, A, m * n * sizeof(Scalar));
//   }
//   printf("MAGMA ran QR on %dx%d matrix in %f s (avg over %d)\n", m, n, magmaElapsed / trials, trials);
// #endif
//   cudaProfilerStop();
//   /*
//   printf("tau values after QR (grid corresponding to columns within panels):\n");
//   for(int j = 0; j < rowPanels; j++)
//   {
//     for(int i = 0; i < colPanels * PC; i++)
//     {
//       printf("%9f ", tau[i * rowPanels + j]);
//     }
//     putchar('\n');
//   }
//   putchar('\n');
//   */
//   //printf("A raw storage after QR:\n");
//   //printMat(RV, m, n);
//   /*
//   Scalar* Q = (Scalar*) malloc(m * m * sizeof(Scalar));
//   Scalar* R = (Scalar*) malloc(m * n * sizeof(Scalar));
//   explicitQR(RV, tau, Q, R, m, n);
//   printf("Q:\n");
//   printMat(Q, m, m);
//   printf("R:\n");
//   printMat(R, m, n);
//   //now compute Q*R explicitly and compare to A
//   Scalar* QR = (Scalar*) malloc(m * n * sizeof(Scalar));
//   dgemm(Q, R, QR, m, m, n);
//   printf("QR:\n");
//   printMat(QR, m, n);
//   Scalar* QRmA = (Scalar*) malloc(m * n * sizeof(Scalar));
//   Scalar errNorm = 0;
//   for(int i = 0; i < m * n; i++)
//   {
//     QRmA[i] = QR[i] - A[i];
//     errNorm += QRmA[i] * QRmA[i];
//   }
//   printf("QR-A (should be 0):\n");
//   printMat(QRmA, m, n);
//   free(QRmA);
//   errNorm = sqrt(errNorm);
//   printf("L2 norm of residual QR-A: %.9g\n", errNorm);
//   free(R);
//   free(Q);
//   free(QR);
//   */
//   free(RV);
//   free(A);
// #ifdef ENABLE_MAGMA
//   magma_finalize();
// #endif
//   return 0;
// }
